#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;

#include <thrust/reduce.h>
#include <thrust/sequence.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

__global__ void fillKernel(int *a,int n)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < n) a[tid] = tid;
}

void fill(int * d_a, int n)
{
    int nThreadPerBlock = 512;
    int nBlocks = n / nThreadPerBlock + ((n%nThreadPerBlock)?1:0);
    fillKernel <<< nBlocks, nThreadPerBlock >>> (d_a, n);
}

int main()
{
    const int N = 50000;
    thrust::device_vector<int> a(N);

    fill(thrust::raw_pointer_cast(&a[0]), N);

    int sumA = thrust::reduce(a.begin(), a.end(), 0);

    int sumCheck = 0;
    for(int i=0; i!=N; ++i)
        sumCheck += i;

    if(sumA == sumCheck)
        cout << "Test Succeeded!" << endl;
    else {
        cerr << "Test FAILED!" << endl;
        return 1;
    }

    return 0;
}